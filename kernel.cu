#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "resource.h"	
#include <stdio.h>
#include <windows.h>
#include <SDKDDKVer.h>

//#include <math_functions.h>
#include <gl/glew.h>
#include <cuda_gl_interop.h>
#include <math.h>

#define MAX_LOADSTRING 100
#define FR 1000.0
#define DT 0.2f 

const int arraySize = 2048;

HINSTANCE hInst;
WCHAR szTitle[MAX_LOADSTRING];
WCHAR szWindowClass[MAX_LOADSTRING];

float *device_a = 0, *device_b = 0, *device_c = 0, *device_d = 0, *device_m = 0;
int *device_img = 0, *device_ct = 0;
hipArray* device_array;
float *partten;
int colortable[4096];
HDC hdc1, hdcc, hdc2;
HGLRC m_hrc;
GLuint pbo, texbuffer;
struct hipGraphicsResource *cuda_pbo_resource;
size_t num_bytes;
int cx, cy;
hipError_t cui(HWND hWnd);
int startt = 0;
unsigned char getr(double x);
unsigned char getg(double x);
unsigned char getb(double x);

ATOM                MyRegisterClass(HINSTANCE hInstance);
BOOL                InitInstance(HINSTANCE, int, HWND*);
LRESULT CALLBACK    WndProc(HWND, UINT, WPARAM, LPARAM);
//INT_PTR CALLBACK    About(HWND, UINT, WPARAM, LPARAM);

__global__ void Kernelevo1(float *a, float *b, float *c, float *d, float *m)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int p = (y+2)*(arraySize+4) + x+2;
	d[p] = b[p] + ((a[p + arraySize + 4] + a[p - arraySize - 4] + a[p + 1] + a[p - 1])*16.0f - a[p + arraySize * 2 + 8] - a[p - arraySize * 2 - 8] - a[p + 2] - a[p - 2] - 60.0f*a[p])*0.08333333333f*DT;
	c[p] = a[p] + (d[p]) * m[p] * DT;
}
__global__ void Kernelevo2(float *a, float *b, float *m) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int p = (y + 2)*(arraySize + 4) + x + 2;
	a[p] += b[p] * m[p] * 0.2f;
}

__global__ void Kernelimg(float *a, int *imgbits, int *ct) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int p = (y + 2)*(arraySize + 4) + x + 2;
	int c = (a[p] + 0.5f) * 2048;
	c = max(0, min(4095, c));
	imgbits[y*arraySize + x] = ct[c];
}

unsigned char getr(double x) {
	return (tanh((x - 0.375) * 8) + 1) * 127;
}
unsigned char getg(double x) {
	return (tanh((x - 0.625) * 8) + 1) * 127;
}
unsigned char getb(double x) {
	return (exp(-25 * (x - 0.28)*(x - 0.25)) *0.5 + 1 + tanh((x - 0.875) * 8)) * 255 / 2;
}

void draw(HDC hdc, HWND hWnd, unsigned int f) {
	/*himg = CreateBitmapIndirect(&img);
	preimg = SelectObject(hdcc, himg);
	BitBlt(hdc, 0, 0, arraySize, arraySize, hdcc, 0, 0, SRCCOPY);
	SelectObject(hdcc, preimg);
	DeleteObject(himg);*/
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, arraySize, arraySize, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	glClear(GL_COLOR_BUFFER_BIT); 
	//glBindTexture(GL_TEXTURE_2D, texbuffer);
	//glCopyPixels(0, 0, arraySize, arraySize, GL_COLOR);
	//glDrawPixels(arraySize, arraySize, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glBegin(GL_QUADS);
	glColor3f(1.0f, 1.0f, 1.0f);
	glTexCoord2f(0.0f, 0.0f);
	glVertex3f(-1.0f, -1.0f, -1.0f);
	glTexCoord2f(1.0f, 0.0f);
	glVertex3f(1.0f, -1.0f, -1.0f);
	glColor3f(1.0f, 1.0f, 0.0f);
	glTexCoord2f(1.0f, 1.0f);
	glVertex3f(1.0f, 1.0f, -1.0f);
	glTexCoord2f(0.0f, 1.0f);
	glVertex3f(-1.0f, 1.0f, -1.0f);
	glEnd();
	SwapBuffers(hdc);
}
hipError_t cui(HWND hWnd) {
	hipError_t cudaStatus;
	int i, j, k;
	double r;
	double r1[10];

	cudaStatus = hipMalloc(&device_a, (arraySize + 4) *(arraySize + 4) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMalloc(&device_b, (arraySize + 4) *(arraySize + 4) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMalloc(&device_c, (arraySize + 4) *(arraySize + 4) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMalloc(&device_d, (arraySize + 4) *(arraySize + 4) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMalloc(&device_m, (arraySize + 4) *(arraySize + 4) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	/*cudaStatus = hipMalloc(&device_img, arraySize*arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "mallloc failed", "message", MB_OK);
		goto Error;
	}*/
	hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
	cudaStatus = hipGraphicsResourceGetMappedPointer((void **)&device_img, &num_bytes, cuda_pbo_resource);
	//cudaStatus = hipGraphicsSubResourceGetMappedArray(&device_array, cuda_pbo_resource,0,0); 
	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "res failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMalloc(&device_ct, 4096 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "malloc failed", "message", MB_OK);
		goto Error;
	}

	partten = (float*)malloc((arraySize + 4) *(arraySize + 4) * sizeof(float));
	srand(GetTickCount());
	for (i = 0; i < arraySize + 4; i++) {
		for (j = 0; j < arraySize + 4; j++) {
			r = ((double)(i - 2) / arraySize - 0.5)*((double)(i - 2) / arraySize - 0.5) + ((double)(j - 2) / arraySize - 0.25)*((double)(j - 2) / arraySize - 0.25);
			//r = sqrt(r);
			partten[i*(arraySize + 4) + j] = (r < 0.06) ? 0.5 + 1.0*(exp(-140.0*r))*cos((double)(j - 2) / arraySize * FR) : 0.5;
		}
	}

	cudaStatus = hipMemcpy(device_a, partten, (arraySize + 4) *(arraySize + 4) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "copy failed", "message", MB_OK);
		goto Error;
	}
	cudaStatus = hipMemcpy(device_c, partten, (arraySize + 4) *(arraySize + 4) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "copy failed", "message", MB_OK);
		goto Error;
	}

	partten = (float*)malloc((arraySize + 4) *(arraySize + 4) * sizeof(float));
	srand(GetTickCount());
	for (i = 0; i < arraySize + 4; i++) {
		for (j = 0; j < arraySize + 4; j++) {
			r = ((double)(i - 2) / arraySize - 0.5)*((double)(i - 2) / arraySize - 0.5) + ((double)(j - 2) / arraySize - 0.25)*((double)(j - 2) / arraySize - 0.25);
			//r = sqrt(r);
			partten[i*(arraySize + 4) + j] = (r < 0.06) ? 1.0*FR / arraySize*(exp(-140.0*r))*sin((double)(j - 2 + DT / 2) / arraySize * FR) : 0.0;
		}
	}

	cudaStatus = hipMemcpy(device_b, partten, (arraySize + 4) *(arraySize + 4) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "copy failed", "message", MB_OK);
		goto Error;
	}

	for (i = 0; i < arraySize + 4; i++) {
		for (j = 0; j < arraySize + 4; j++) {
			partten[i*(arraySize + 4) + j] = 1;
			for (k = 0; k < 61; k++) {
				r = ((double)(i - 2) / arraySize - k*0.01 - 0.2)*((double)(i - 2) / arraySize - k*0.01 - 0.2) + ((double)(j - 2) / arraySize - k*0.003 - 0.41)*((double)(j - 2) / arraySize - k*0.003 - 0.41);
				if (r < 0.000005) {
					partten[i*(arraySize + 4) + j] = 0;
					break;
				}
			}
		}
	}

	cudaStatus = hipMemcpy(device_m, partten, (arraySize + 4) *(arraySize + 4) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "copy failed", "message", MB_OK);
		goto Error;
	}

	cudaStatus = hipMemcpy(device_ct, colortable, 4096 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		MessageBoxA(hWnd, "copy failed", "message", MB_OK);
		goto Error;
	}
	
	return cudaStatus;

Error:
	hipFree(device_a);
	hipFree(device_b);
	return cudaStatus;
}

int APIENTRY wWinMain(_In_ HINSTANCE hInstance, _In_opt_ HINSTANCE hPrevInstance, _In_ LPWSTR lpCmdLine, _In_ int nCmdShow) {
	MSG msg;
	HWND hWnd;
	int i, j;
	unsigned int t, t1, t2, count, f;
	hipError_t cudaStatus;
	dim3 blocksize, gridsize;
	char s[32];
	LoadString(hInstance, IDS_APP_TITLE, szTitle, MAX_LOADSTRING);
	LoadString(hInstance, IDS_WIN32PROJECT1, szWindowClass, MAX_LOADSTRING);
	MyRegisterClass(hInstance);
	if (!InitInstance(hInstance, nCmdShow, &hWnd))
	{
		return FALSE;
	}
	blocksize = dim3(128, 1, 1);
	gridsize = dim3(arraySize / blocksize.x, arraySize / blocksize.y, 1);
	t = GetTickCount();
	count = 0;
	f = 0;
	for (;;) {
		if (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE))
		{
			if (msg.message == WM_QUIT)
				break;
			TranslateMessage(&msg);
			DispatchMessage(&msg);
		}
		else if (startt) {
			for (i = 0; i < startt; i++) {
				Kernelevo1 << <gridsize, blocksize >> > (device_a, device_b, device_c, device_d, device_m);
				Kernelevo1 << <gridsize, blocksize >> > (device_c, device_d, device_a, device_b, device_m);
			}
			//cudaStatus = hipGraphicsMapResources(1, &cuda_pbo_resource, 0); 
			/*if (cudaStatus != hipSuccess) {
				MessageBoxA(hWnd, "evo failed", "message", MB_OK);
				break;
			}*/
			Kernelimg << <gridsize, blocksize >> > (device_a, device_img, device_ct);
			//cudaStatus = hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
			cudaStatus = cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				MessageBoxA(hWnd, "evo failed", "message", MB_OK);
				break;
			}
			draw(hdc1, hWnd, f);
			count += 1;
			t1 = GetTickCount();
			if (t1 - t > 1000) {
				t += 1000;
				sprintf(s, "%dfps", count);
				TextOutA(hdc2, 0, 0, s, strlen(s));
				count = 0;
			}
		}
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}

ATOM MyRegisterClass(HINSTANCE hInstance)
{
	WNDCLASSEXW wcex;

	wcex.cbSize = sizeof(WNDCLASSEX);

	wcex.style = CS_HREDRAW | CS_VREDRAW;
	wcex.lpfnWndProc = WndProc;
	wcex.cbClsExtra = 0;
	wcex.cbWndExtra = 0;
	wcex.hInstance = hInstance;
	wcex.hIcon = LoadIcon(hInstance, MAKEINTRESOURCE(IDI_ICON1));
	wcex.hCursor = LoadCursor(nullptr, IDC_ARROW);
	wcex.hbrBackground = (HBRUSH)(COLOR_WINDOW + 1);
	wcex.lpszMenuName = MAKEINTRESOURCE(IDR_MENU1);
	wcex.lpszClassName = szWindowClass;
	wcex.hIconSm = LoadIcon(wcex.hInstance, MAKEINTRESOURCE(IDI_ICON1));
	return RegisterClassEx(&wcex);
}

BOOL InitInstance(HINSTANCE hInstance, int nCmdShow, HWND* hWnd)
{
	hInst = hInstance; // ��ʵ������洢��ȫ�ֱ�����

	*hWnd = CreateWindow(szWindowClass, szTitle, WS_OVERLAPPEDWINDOW, 0, 0, 1060, 1040, nullptr, nullptr, hInstance, nullptr);

	if (!hWnd)
	{
		return FALSE;
	}

	ShowWindow(*hWnd, nCmdShow);
	UpdateWindow(*hWnd);

	return TRUE;
}


LRESULT CALLBACK WndProc(HWND hWnd, UINT message, WPARAM wParam, LPARAM lParam)
{
	switch (message)
	{
	case WM_COMMAND:
	{
		int wmId = LOWORD(wParam);
		switch (wmId) {
		default:
			return DefWindowProc(hWnd, message, wParam, lParam);
		}
	}
	break;
	case WM_PAINT: {
		PAINTSTRUCT ps;
		HDC hdc = BeginPaint(hWnd, &ps);
		draw(hdc1, hWnd, 0);
		EndPaint(hWnd, &ps);
		break;
	}
	case WM_CREATE: {
		hipError_t cudaStatus;
		SetTimer(hWnd, 1, 10, NULL);
		hdc1 = GetDC(hWnd);
		hdc2 = GetDC(NULL);
		hdcc = CreateCompatibleDC(hdc1);

		PIXELFORMATDESCRIPTOR pfd = {
			sizeof(PIXELFORMATDESCRIPTOR),
			1,
			PFD_DRAW_TO_WINDOW | PFD_SUPPORT_OPENGL | PFD_DOUBLEBUFFER | PFD_STEREO,
			PFD_TYPE_RGBA,
			24,
			0,0,0,0,0,0,0,0,
			0,
			0,0,0,0,
			32,
			0,0,
			PFD_MAIN_PLANE,
			0,0,0,0
		};
		int uds = ::ChoosePixelFormat(hdc1, &pfd);
		::SetPixelFormat(hdc1, uds, &pfd);
		m_hrc = ::wglCreateContext(hdc1);
		::wglMakeCurrent(hdc1, m_hrc);
		glewInit();
		glDisable(GL_DEPTH_TEST);
		glEnable(GL_TEXTURE_2D);
		glClearColor(1.0, 1.0, 1.0, 1.0); 

		glMatrixMode(GL_PROJECTION);
		glLoadIdentity();
		glFrustum(-0.5, 0.5, -0.5, 0.5, 0.5, 2.0);
		glMatrixMode(GL_MODELVIEW);
		glLoadIdentity();

		glGenBuffers(1, &pbo);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
		glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, arraySize*arraySize * 4, 0, GL_STREAM_DRAW_ARB);

		glGenTextures(1, &texbuffer);
		glBindTexture(GL_TEXTURE_2D, texbuffer);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
		//glBindTexture(GL_TEXTURE_2D, 0);

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			MessageBoxA(hWnd, "device failed", "message", MB_OK);
			PostQuitMessage(1);
		}
		
		cudaStatus = hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, texbuffer, cudaGraphicsMapFlagsWriteDiscard);
		//cudaStatus = hipGraphicsGLRegisterImage(&cuda_pbo_resource, texbuffer, GL_TEXTURE_2D, cudaGraphicsMapFlagsWriteDiscard);
		if (cudaStatus != hipSuccess) {
			MessageBoxA(hWnd, "pbo failed", "message", MB_OK);
			PostQuitMessage(1);
		}
		for (int i = 0; i < 4096; i++) {
			colortable[i] = (255 << 24) + (getb((double)i / 2048 - 0.5) << 16) + (getg((double)i / 2048 - 0.5) << 8) + getr((double)i / 2048 - 0.5);
		}
		cudaStatus = cui(hWnd);
		if (cudaStatus != hipSuccess) {
			PostQuitMessage(1);
		}
		Kernelimg << <dim3(arraySize / 128, arraySize / 1), dim3(128, 1) >> > (device_a, device_img, device_ct);
		cudaStatus = cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			MessageBoxA(hWnd, "evo failed", "message", MB_OK);
			PostQuitMessage(1);
		}

		break;
	}
	case WM_TIMER: {
		break;
	}
	case WM_SIZE: {
		cx = lParam & 0xffff;
		cy = (lParam & 0xffff0000) >> 16;
		if (cx <= cy) {
			glViewport(0, (cy - cx) / 2, cx, cx);
		}
		else {
			glViewport((cx - cy) / 2, 0, cy, cy);
		}
		break;
	}
	case WM_KEYDOWN: {
		switch (wParam) {
		case ' ': {
			startt = 10 - startt;
			break;
		}
		}
		break;
	}
	case WM_DESTROY:
		PostQuitMessage(0);
		break;
	default:
		return DefWindowProc(hWnd, message, wParam, lParam);
	}
	return 0;
}
//glew32.lib;glu32.lib;opengl32.lib;