#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "mimg.h"
#include <cuda_gl_interop.h>


const float pi = 3.1415926536f;
const int size = 2048;
double2 *device_p1 = NULL;
int *device_img = NULL, *device_ct = NULL;
char *pattern = NULL;
struct hipGraphicsResource *cuda_vbo_resource;
int colortable[256];

unsigned char getr(double x) {
	return (tanh((x - 0.4) * 8) + 2 + tanh((0.2 - x) * 10) - exp(-(x - 1.0)*(x - 1.0) * 100)*0.05) * 127;
}
unsigned char getg(double x) {
	return (tanh((x - 0.70) * 8) + 2 + tanh((0.2 - x) * 10) - exp(-(x - 1.0)*(x - 1.0) * 100)*0.0) * 127;
}
unsigned char getb(double x) {
	return (tanh((0.45 - x) * 8) + 2.0 + tanh((x - 0.92) * 10) + exp(-(x - 1.0)*(x - 1.0) * 100)*0.2) * 255 / 2;
}


__global__ void evo(double2 *p1, int *img,int *ct, double left, double bottom, double d) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int i = x + y*size;
	int j;
	double2 a, b, c;
	a.x = left + d*x;
	a.y = bottom + d*y;
	b = p1[i];
	c.x = b.x*b.x;
	c.y = b.y*b.y;
	if (c.x + c.y > 4.0) {
		return;
	}
	b.y *= b.x;
	b.y += b.y + a.y;
	b.x = c.x - c.y + a.x;
	for (j = 0; j < 255; j++) {
		c.x = b.x*b.x;
		c.y = b.y*b.y;
		if (c.x + c.y > 4.0) {
			img[i] = ct[j];
			p1[i] = b;
			return;
		}
		b.y *= b.x;
		b.y += b.y + a.y;
		b.x = c.x - c.y + a.x;
	}
	c.x = b.x*b.x;
	c.y = b.y*b.y;
	if (c.x + c.y > 4.0) {
		img[i] = ct[255];
	}
	p1[i] = b;
}
__global__ void clear(double2 *p1, int *img) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int i = x + y*size;
	p1[i].x = 0.0;
	p1[i].y = 0.0;
	img[i] = 0;
}
int cudainit(GLuint pbo) {
	hipError_t cudaStatus;
	size_t num_bytes;
	int i;
	float x;
	pattern = (char*)malloc(size*size * 4);
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMalloc(&device_p1, size*size * sizeof(double2));
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMalloc(&device_ct, 1024 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	for (i = 0; i < 256; i++) {
		colortable[i] = (255 << 24) + (getb((double)i / 256) << 16) + (getg((double)i / 256) << 8) + getr((double)i / 256);
	}

	cudaStatus = hipMemcpy(device_ct, colortable, 256 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	cudaStatus = hipGraphicsResourceGetMappedPointer((void **)&device_img, &num_bytes, cuda_vbo_resource);
	hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	return cudaimginit();
Error:
	hipFree(device_p1);
	return 1;
}

int cudacalc(double left, double bottom, double d) {
	hipError_t cudaStatus;
	//hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	evo << <dim3(256, 135), dim3(8, 8) >> > (device_p1, device_img,device_ct, left, bottom, d);
	cudaStatus = hipDeviceSynchronize();
	//hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}

int cudafin(void) {
	hipError_t cudaStatus;
	hipFree(device_p1);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}


int cudaimginit(void) {
	hipError_t cudaStatus;
	clear << <dim3(16, 2048), dim3(128, 1) >> > (device_p1, device_img);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		return 1;
	}
	return 0;
}